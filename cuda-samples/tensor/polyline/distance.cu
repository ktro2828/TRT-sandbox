
#include <hip/hip_runtime.h>
#include <iostream>

/**
 * @brief Calculate center distance from target agent to each polyline.
 *
 * @note Polyline must have been transformed to target agent coordinates system.
 *
 * @param B The number of target agents.
 * @param K The number of polylines.
 * @param P The number of points contained in each polyline.
 * @param D The number of point dimensions, expecting [x, y, ...].
 * @param polyline Source polyline, in shape [B*K*P*D].
 * @param polylineMask Source polyline mask, in shape [B*K*P].
 * @param distance Output calculated distances, in shape [B*K].
 */
__global__ void calculateCenterDistanceKernel(
  const int B, const int K, const int P, const int D, const float * polyline,
  const bool * polylineMask, float * distance)
{
  int b = blockIdx.x * blockDim.x + threadIdx.x;
  int k = blockIdx.y * blockDim.y + threadIdx.y;
  if (b >= B || k >= K) {
    return;
  }

  // calculate polyline center
  float sumX = 0.0f, sumY = 0.0f;
  int numValid = 0;
  for (int p = 0; p < P; ++p) {
    int idx = b * K * P + k * P + p;
    if (polylineMask[idx]) {
      sumX += polyline[idx * D];
      sumY += polyline[idx * D + 1];
      ++numValid;
    }
  }
  float centerX = sumX / fmaxf(1.0f, numValid);
  float centerY = sumY / fmaxf(1.0f, numValid);

  distance[b * K + k] = hypot(centerX, centerY);
}

int main()
{
  constexpr int B = 2;
  constexpr int K = 3;
  constexpr int P = 4;
  constexpr int D = 9;  // (x, y, z, dx, dy, dz, typeID, preX, preY)

  float h_polyline[B][K][P][D] = {
    {{
       {1.0f, 1.5f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f, 0.0f},
       {2.0f, 2.5f, 2.0f, 2.0f, 2.0f, 2.0f, 1.0f, 0.0f, 0.0f},
       {3.0f, 3.5f, 3.0f, 3.0f, 3.0f, 3.0f, 1.0f, 0.0f, 0.0f},
       {4.0f, 4.5f, 4.0f, 4.0f, 4.0f, 4.0f, 1.0f, 0.0f, 0.0f},
     },
     {
       {2.0f, 2.5f, 2.0f, 2.0f, 2.0f, 2.0f, 1.0f, 0.0f, 0.0f},
       {3.0f, 3.5f, 3.0f, 3.0f, 3.0f, 3.0f, 1.0f, 0.0f, 0.0f},
       {4.0f, 4.5f, 4.0f, 4.0f, 4.0f, 4.0f, 1.0f, 0.0f, 0.0f},
       {5.0f, 5.5f, 5.0f, 5.0f, 5.0f, 5.0f, 1.0f, 0.0f, 0.0f},
     },
     {
       {3.0f, 3.5f, 3.0f, 3.0f, 3.0f, 3.0f, 1.0f, 0.0f, 0.0f},
       {4.0f, 4.5f, 4.0f, 4.0f, 4.0f, 4.0f, 1.0f, 0.0f, 0.0f},
       {5.0f, 5.5f, 5.0f, 5.0f, 5.0f, 5.0f, 1.0f, 0.0f, 0.0f},
       {5.0f, 5.5f, 5.0f, 5.0f, 5.0f, 5.0f, 1.0f, 0.0f, 0.0f},
     }},
    {{
       {1.0f, 1.5f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f, 0.0f},
       {2.0f, 2.5f, 2.0f, 2.0f, 2.0f, 2.0f, 1.0f, 0.0f, 0.0f},
       {3.0f, 3.5f, 3.0f, 3.0f, 3.0f, 3.0f, 1.0f, 0.0f, 0.0f},
       {5.0f, 5.5f, 5.0f, 5.0f, 5.0f, 5.0f, 1.0f, 0.0f, 0.0f},
     },
     {
       {1.0f, 1.5f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f, 0.0f},
       {2.0f, 2.5f, 2.0f, 2.0f, 2.0f, 2.0f, 1.0f, 0.0f, 0.0f},
       {3.0f, 3.5f, 3.0f, 3.0f, 3.0f, 3.0f, 1.0f, 0.0f, 0.0f},
       {5.0f, 5.5f, 5.0f, 5.0f, 5.0f, 5.0f, 1.0f, 0.0f, 0.0f},
     },
     {
       {1.0f, 1.5f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f, 0.0f},
       {2.0f, 2.5f, 2.0f, 2.0f, 2.0f, 2.0f, 1.0f, 0.0f, 0.0f},
       {3.0f, 3.5f, 3.0f, 3.0f, 3.0f, 3.0f, 1.0f, 0.0f, 0.0f},
       {4.0f, 4.5f, 4.0f, 4.0f, 4.0f, 4.0f, 1.0f, 0.0f, 0.0f},
     }}};

  bool h_polylineMask[B][K][P] = {
    {{true, true, true, true}, {true, true, true, true}, {true, true, true, true}},
    {{true, true, true, true}, {true, true, true, true}, {true, true, true, true}}};

  const size_t polylineNBytes = sizeof(float) * B * K * P * D;
  const size_t polylineMaskNBytes = sizeof(bool) * B * K * P;
  const size_t distanceNBytes = sizeof(float) * B * K;

  float *d_polyline, *d_distance;
  bool * d_polylineMask;
  hipMalloc(&d_polyline, polylineNBytes);
  hipMalloc(&d_polylineMask, polylineMaskNBytes);
  hipMemcpy(d_polyline, h_polyline, polylineNBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_polylineMask, h_polylineMask, polylineMaskNBytes, hipMemcpyHostToDevice);

  hipMalloc(&d_distance, distanceNBytes);

  dim3 blocks(B, K);
  constexpr int THREADS_PER_BLOCK = 256;
  calculateCenterDistanceKernel<<<blocks, THREADS_PER_BLOCK>>>(
    B, K, P, D, d_polyline, d_polylineMask, d_distance);

  auto err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
  }

  float h_distance[B][K];
  hipMemcpy(h_distance, d_distance, distanceNBytes, hipMemcpyDeviceToHost);

  std::cout << "=== Out distance ===\n";
  for (int b = 0; b < B; ++b) {
    std::cout << "Batch " << b << ": ";
    for (int k = 0; k < K; ++k) {
      std::cout << h_distance[b][k] << " ";
    }
    std::cout << "\n";
  }

  hipFree(d_polyline);
  hipFree(d_polylineMask);
  hipFree(d_distance);
}