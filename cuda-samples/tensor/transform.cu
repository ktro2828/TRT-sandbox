
#include <hip/hip_runtime.h>
#include <iostream>

/**
 * @brief Transform the trajectory coords to the coords system centered around the target object.
 *
 * @param B
 * @param N
 * @param T
 * @param D
 * @param in_trajectory
 * @param center_xyz
 * @param center_yaw
 * @param output
 */
__global__ void transform_trajectory_kernel(
  const int B, const int N, const int T, const int D, const float * in_trajectory,
  const float * center_xyz, const float * center_yaw, float * output)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N * T) {
    const float x = in_trajectory[idx * D];
    const float y = in_trajectory[idx * D + 1];
    const float z = in_trajectory[idx * D + 2];
    const float dx = in_trajectory[idx * D + 3];
    const float dy = in_trajectory[idx * D + 4];
    const float dz = in_trajectory[idx * D + 5];
    const float yaw = in_trajectory[idx * D + 6];
    const float vx = in_trajectory[idx * D + 7];
    const float vy = in_trajectory[idx * D + 8];
    const float is_valid = in_trajectory[idx * D + 9];

    // transform for each target
    for (int b = 0; b < B; ++b) {
      const float center_x = center_xyz[b * 3];
      const float center_y = center_xyz[b * 3 + 1];
      const float center_z = center_xyz[b * 3 + 2];
      const float cos_val = std::cos(center_yaw[b]);
      const float sin_val = std::sin(center_yaw[b]);

      // transform
      const float trans_x = cos_val * (x - center_x) - sin_val * (y - center_y);
      const float trans_y = sin_val * (x - center_x) + cos_val * (y - center_y);
      const float trans_z = z - center_z;
      const float trans_yaw = yaw - center_yaw[b];
      const float trans_vx = cos_val * vx - sin_val * vy;
      const float trans_vy = sin_val * vx + cos_val * vy;

      const int trans_idx = (b * N * T + idx) * D;
      output[trans_idx] = trans_x;
      output[trans_idx + 1] = trans_y;
      output[trans_idx + 2] = trans_z;
      output[trans_idx + 3] = dx;
      output[trans_idx + 4] = dy;
      output[trans_idx + 5] = dz;
      output[trans_idx + 6] = trans_yaw;
      output[trans_idx + 7] = trans_vx;
      output[trans_idx + 8] = trans_vy;
      output[trans_idx + 9] = is_valid;
    }
  }
}

__global__ void extract_last_pos_kernel(
  const int B, const int N, const int T, const int D, const float * in_trajectory, float * output)
{
  int b = blockIdx.x * blockDim.x + threadIdx.x;
  int n = blockIdx.y * blockDim.y + threadIdx.y;
  int t = blockIdx.z * blockDim.z + threadIdx.z;
  if (b < B && t == T - 1) {
    const int idx = b * N * T + n * T + t;
    const int out_idx = b * N + n;
    output[out_idx] = 0.0f;
    output[out_idx * 3] = in_trajectory[idx * D];
    output[out_idx * 3 + 1] = in_trajectory[idx * D + 1];
    output[out_idx * 3 + 2] = in_trajectory[idx * D + 2];
  }
}

int main()
{
  constexpr int B = 2;   // Batch size
  constexpr int N = 3;   // The number of agents
  constexpr int T = 5;   // The number of timestamps
  constexpr int D = 10;  // The number of state dimensions
  float h_src[N][T][D] = {
    {{1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f},
     {1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 1.0f},
     {1.0f, 1.0f, 1.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 1.0f}},
    {{2.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {2.0f, 2.0f, 2.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f}},
    {{2.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {1.0f, 2.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f},
     {3.0f, 3.0f, 3.0f, 0.1f, 0.2f, 1.0f, 2.0f, 3.0f, 0.1f, 0.0f}}};

  float h_xyz[B][3] = {{1.0f, 1.0f, 1.0f}, {2.0f, 2.0f, 2.0f}};  // (x, y, z)[m]
  float h_yaw[B] = {1.0f, 0.0f};                                 // [deg]

  float *d_src, *d_xyz, *d_yaw, *d_dst;
  const size_t in_size = sizeof(float) * N * T * D;
  const size_t xyz_size = sizeof(float) * B * 3;
  const size_t yaw_size = sizeof(float) * B;
  const size_t out_size = sizeof(float) * B * N * T * D;
  hipMalloc(reinterpret_cast<void **>(&d_src), in_size);
  hipMalloc(reinterpret_cast<void **>(&d_xyz), xyz_size);
  hipMalloc(reinterpret_cast<void **>(&d_yaw), yaw_size);
  hipMalloc(reinterpret_cast<void **>(&d_dst), out_size);
  hipMemcpy(d_src, h_src, in_size, hipMemcpyHostToDevice);
  hipMemcpy(d_xyz, h_xyz, xyz_size, hipMemcpyHostToDevice);
  hipMemcpy(d_yaw, h_yaw, yaw_size, hipMemcpyHostToDevice);

  dim3 blocks(N, T);
  transform_trajectory_kernel<<<blocks, 256>>>(B, N, T, D, d_src, d_xyz, d_yaw, d_dst);

  float h_dst[B][N][T][D];
  hipMemcpy(h_dst, d_dst, out_size, hipMemcpyDeviceToHost);

  std::cout << "Transform coords to the target centric coords..." << std::endl;
  for (int b = 0; b < B; ++b) {
    std::cout << "Batch " << b << ":\n";
    for (int n = 0; n < N; ++n) {
      std::cout << "  Agent " << n << ":\n";
      for (int t = 0; t < T; ++t) {
        std::cout << "  Time " << t << ": ";
        for (int i = 0; i < D; ++i) {
          std::cout << h_dst[b][n][t][i] << " ";
        }
        std::cout << "\n";
      }
    }
  }

  float * d_last_pos;
  hipMalloc(reinterpret_cast<void **>(&d_last_pos), sizeof(float) * B * N * 3);
  dim3 nBlocks(B, N, T);
  extract_last_pos_kernel<<<nBlocks, 256>>>(B, N, T, D, d_dst, d_last_pos);

  float h_last_pos[B][N][3];
  hipMemcpy(h_last_pos, d_last_pos, sizeof(float) * B * N * 3, hipMemcpyDeviceToHost);

  std::cout << "Extract last positions..." << std::endl;
  for (int b = 0; b < B; ++b) {
    std::cout << "Batch " << b << ":\n";
    for (int n = 0; n < N; ++n) {
      std::cout << "  Agent " << n << ": ";
      for (int i = 0; i < 3; ++i) {
        std::cout << h_last_pos[b][n][i] << " ";
      }
      std::cout << "\n";
    }
  }

  hipFree(d_src);
  hipFree(d_xyz);
  hipFree(d_yaw);
  hipFree(d_dst);
  hipFree(d_last_pos);
}
