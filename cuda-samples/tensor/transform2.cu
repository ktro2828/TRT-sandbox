
#include <hip/hip_runtime.h>
#include <iostream>

/**
 * @brief Transform the trajectory coords to the coords system centered around the target object.
 *
 * @param B
 * @param N
 * @param T
 * @param D
 * @param in_trajectory
 * @param target_index
 * @param output
 */
__global__ void transform_trajectory_kernel(
  const int B, const int N, const int T, const int D, const float * in_trajectory,
  const int * target_index, float * output)
{
  int b = blockIdx.x * blockDim.x + threadIdx.x;
  int n = blockIdx.y * blockDim.y + threadIdx.y;
  int t = blockIdx.z * blockDim.z + threadIdx.z;
  if (b < B && n < N && t < T) {
    int src_idx = n * T + t;
    const float x = in_trajectory[src_idx * D];
    const float y = in_trajectory[src_idx * D + 1];
    const float z = in_trajectory[src_idx * D + 2];
    const float dx = in_trajectory[src_idx * D + 3];
    const float dy = in_trajectory[src_idx * D + 4];
    const float dz = in_trajectory[src_idx * D + 5];
    const float yaw = in_trajectory[src_idx * D + 6];
    const float vx = in_trajectory[src_idx * D + 7];
    const float vy = in_trajectory[src_idx * D + 8];
    const float ax = in_trajectory[src_idx * D + 9];
    const float ay = in_trajectory[src_idx * D + 10];
    const float is_valid = in_trajectory[src_idx * D + 11];

    // transform for each target
    const int tgt_idx = (target_index[b] * T + T - 1) * D;

    const float tgt_x = in_trajectory[tgt_idx];
    const float tgt_y = in_trajectory[tgt_idx + 1];
    const float tgt_z = in_trajectory[tgt_idx + 2];
    const float tgt_yaw = in_trajectory[tgt_idx + 6];
    const float cos_val = cos(tgt_yaw);
    const float sin_val = sin(tgt_yaw);

    // transform
    const float trans_x = cos_val * (x - tgt_x) - sin_val * (y - tgt_y);
    const float trans_y = sin_val * (x - tgt_x) + cos_val * (y - tgt_y);
    const float trans_z = z - tgt_z;
    const float trans_yaw = yaw - tgt_yaw;
    const float trans_vx = cos_val * vx - sin_val * vy;
    const float trans_vy = sin_val * vx + cos_val * vy;
    const float trans_ax = cos_val * ax - sin_val * ay;
    const float trans_ay = sin_val * ax + cos_val * ay;

    const int trans_idx = (b * N * T + n * T + t) * D;
    output[trans_idx] = trans_x;
    output[trans_idx + 1] = trans_y;
    output[trans_idx + 2] = trans_z;
    output[trans_idx + 3] = dx;
    output[trans_idx + 4] = dy;
    output[trans_idx + 5] = dz;
    output[trans_idx + 6] = trans_yaw;
    output[trans_idx + 7] = trans_vx;
    output[trans_idx + 8] = trans_vy;
    output[trans_idx + 9] = trans_ax;
    output[trans_idx + 10] = trans_ay;
    output[trans_idx + 11] = is_valid;
  }
}

__global__ void extract_last_pos_kernel(
  const int B, const int N, const int T, const int D, const float * in_trajectory, float * output)
{
  int b = blockIdx.x * blockDim.x + threadIdx.x;
  int n = blockIdx.y * blockDim.y + threadIdx.y;
  int t = blockIdx.z * blockDim.z + threadIdx.z;
  if (b < B && t == T - 1) {
    const int idx = b * N * T + n * T + t;
    const int out_idx = b * N + n;
    output[out_idx] = 0.0f;
    output[out_idx * 3] = in_trajectory[idx * D];
    output[out_idx * 3 + 1] = in_trajectory[idx * D + 1];
    output[out_idx * 3 + 2] = in_trajectory[idx * D + 2];
  }
}

int main()
{
  constexpr int B = 2;   // Batch size
  constexpr int N = 4;   // The number of agents
  constexpr int T = 5;   // The number of timestamps
  constexpr int D = 12;  // The number of state dimensions
  float h_src[N][T][D] = {
    {
      {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, M_PI / 2, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f},
      {2.0f, 2.0f, 2.0f, 2.0f, 2.0f, 2.0f, M_PI / 2, 2.0f, 2.0f, 2.0f, 2.0f, 1.0f},
      {3.0f, 3.0f, 3.0f, 3.0f, 3.0f, 3.0f, M_PI / 2, 3.0f, 3.0f, 3.0f, 3.0f, 1.0f},
      {4.0f, 4.0f, 4.0f, 4.0f, 4.0f, 4.0f, M_PI / 2, 4.0f, 4.0f, 4.0f, 4.0f, 1.0f},
      {5.0f, 5.0f, 5.0f, 5.0f, 5.0f, 5.0f, M_PI / 2, 5.0f, 5.0f, 5.0f, 5.0f, 1.0f},
    },
    {
      {2.0f, 2.0f, 2.0f, 2.0f, 2.0f, 2.0f, M_PI / 2, 2.0f, 2.0f, 2.0f, 2.0f, 0.0f},
      {3.0f, 3.0f, 3.0f, 3.0f, 3.0f, 3.0f, M_PI / 2, 3.0f, 3.0f, 3.0f, 3.0f, 0.0f},
      {4.0f, 4.0f, 4.0f, 4.0f, 4.0f, 4.0f, M_PI / 2, 4.0f, 4.0f, 4.0f, 4.0f, 1.0f},
      {5.0f, 5.0f, 5.0f, 5.0f, 5.0f, 5.0f, M_PI / 2, 5.0f, 5.0f, 5.0f, 5.0f, 0.0f},
      {6.0f, 6.0f, 6.0f, 6.0f, 6.0f, 6.0f, M_PI / 2, 6.0f, 6.0f, 6.0f, 6.0f, 1.0f},
    },
    {
      {3.0f, 3.0f, 3.0f, 3.0f, 3.0f, 3.0f, M_PI / 2, 3.0f, 3.0f, 3.0f, 3.0f, 1.0f},
      {4.0f, 4.0f, 4.0f, 4.0f, 4.0f, 4.0f, M_PI / 2, 4.0f, 4.0f, 4.0f, 4.0f, 1.0f},
      {5.0f, 5.0f, 5.0f, 5.0f, 5.0f, 5.0f, M_PI / 2, 5.0f, 5.0f, 5.0f, 5.0f, 1.0f},
      {6.0f, 6.0f, 6.0f, 6.0f, 6.0f, 6.0f, M_PI / 2, 6.0f, 6.0f, 6.0f, 6.0f, 1.0f},
      {7.0f, 7.0f, 7.0f, 7.0f, 7.0f, 7.0f, M_PI / 2, 7.0f, 7.0f, 7.0f, 7.0f, 1.0f},
    },
    {
      {4.0f, 4.0f, 4.0f, 4.0f, 4.0f, 4.0f, M_PI / 2, 4.0f, 4.0f, 4.0f, 4.0f, 0.0f},
      {5.0f, 5.0f, 5.0f, 5.0f, 5.0f, 5.0f, M_PI / 2, 5.0f, 5.0f, 5.0f, 5.0f, 1.0f},
      {6.0f, 6.0f, 6.0f, 6.0f, 6.0f, 6.0f, M_PI / 2, 6.0f, 6.0f, 6.0f, 6.0f, 0.0f},
      {7.0f, 7.0f, 7.0f, 7.0f, 7.0f, 7.0f, M_PI / 2, 7.0f, 7.0f, 7.0f, 7.0f, 0.0f},
      {8.0f, 8.0f, 8.0f, 8.0f, 8.0f, 8.0f, M_PI / 2, 8.0f, 8.0f, 8.0f, 8.0f, 1.0f},
    },
  };
  int h_target_index[B] = {0, 2};

  float *d_src, *d_dst;
  int * d_target_index;
  const size_t in_size = sizeof(float) * N * T * D;
  const size_t out_size = sizeof(float) * B * N * T * D;
  hipMalloc(reinterpret_cast<void **>(&d_src), in_size);
  hipMalloc(reinterpret_cast<void **>(&d_target_index), sizeof(int) * B);
  hipMalloc(reinterpret_cast<void **>(&d_dst), out_size);
  hipMemcpy(d_src, h_src, in_size, hipMemcpyHostToDevice);
  hipMemcpy(d_target_index, h_target_index, sizeof(int) * B, hipMemcpyHostToDevice);

  dim3 blocks(B, N, T);
  transform_trajectory_kernel<<<blocks, 1023>>>(B, N, T, D, d_src, d_target_index, d_dst);

  float h_dst[B][N][T][D];
  hipMemcpy(h_dst, d_dst, out_size, hipMemcpyDeviceToHost);

  std::cout << "Transform coords to the target centric coords..." << std::endl;
  for (int b = 0; b < B; ++b) {
    std::cout << "Batch " << b << ":\n";
    for (int n = 0; n < N; ++n) {
      std::cout << "  Agent " << n << ":\n";
      for (int t = 0; t < T; ++t) {
        std::cout << "  Time " << t << ": ";
        for (int i = 0; i < D; ++i) {
          std::cout << h_dst[b][n][t][i] << " ";
        }
        std::cout << "\n";
      }
    }
  }

  float * d_last_pos;
  hipMalloc(reinterpret_cast<void **>(&d_last_pos), sizeof(float) * B * N * 3);
  dim3 nBlocks(B, N, T);
  extract_last_pos_kernel<<<nBlocks, 256>>>(B, N, T, D, d_dst, d_last_pos);

  float h_last_pos[B][N][3];
  hipMemcpy(h_last_pos, d_last_pos, sizeof(float) * B * N * 3, hipMemcpyDeviceToHost);

  std::cout << "Extract last positions..." << std::endl;
  for (int b = 0; b < B; ++b) {
    std::cout << "Batch " << b << ":\n";
    for (int n = 0; n < N; ++n) {
      std::cout << "  Agent " << n << ": ";
      for (int i = 0; i < 3; ++i) {
        std::cout << h_last_pos[b][n][i] << " ";
      }
      std::cout << "\n";
    }
  }

  hipFree(d_src);
  hipFree(d_target_index);
  hipFree(d_dst);
  hipFree(d_last_pos);
}
