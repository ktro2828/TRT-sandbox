#include <hipcub/hipcub.hpp>

#include <iostream>
#include <vector>

/**
 * hipcub::DeviceRadixSort::SortPairsDescending(...)
 *
 * Sorts key-value pairs into descending order. (~2N auxiliary storage required)
 */

template <typename T>
std::ostream & operator<<(std::ostream & os, const std::vector<T> & v)
{
  os << "[";
  for (const auto & e : v) {
    os << e << ", ";
  }
  os << "]";
  return os;
}

int main()
{
  std::vector<float> h_keys{8.0f, 6.0f, 7.0f, 5.0f, 3.0f, 0.0f, 9.0f};
  std::vector<float> h_values{0.0f, 1.0f, 2.0f, 3.0f, 4.0f, 5.0, 6.0f};

  std::cout << "[Before]:\n";
  std::cout << "keys: ";
  std::cout << h_keys << std::endl;
  std::cout << "values: ";
  std::cout << h_values << std::endl;

  size_t num_items = h_keys.size();
  float *d_keys_in, *d_keys_out;
  float *d_values_in, *d_values_out;

  hipMalloc(&d_keys_in, sizeof(float) * num_items);
  hipMalloc(&d_keys_out, sizeof(float) * num_items);
  hipMalloc(&d_values_in, sizeof(float) * num_items);
  hipMalloc(&d_values_out, sizeof(float) * num_items);

  hipMemcpy(d_keys_in, h_keys.data(), sizeof(float) * num_items, hipMemcpyHostToDevice);
  hipMemcpy(d_values_in, h_values.data(), sizeof(float) * num_items, hipMemcpyHostToDevice);

  // Determinate temporary device storage requirements
  void * d_temp_storage{nullptr};
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending(
    d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out,
    num_items);

  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairsDescending(
    d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out,
    num_items);

  std::vector<float> h_keys_out(num_items);
  std::vector<float> h_values_out(num_items);
  hipMemcpy(h_keys_out.data(), d_keys_out, sizeof(float) * num_items, hipMemcpyDeviceToHost);
  hipMemcpy(h_values_out.data(), d_values_out, sizeof(float) * num_items, hipMemcpyDeviceToHost);

  // keys: [9, 8, 7, 6, 5, 3, 0]
  // values: [6, 0, 2, 1, 3, 4, 5]
  std::cout << "[After]:\n";
  std::cout << "keys: ";
  std::cout << h_keys_out << std::endl;
  std::cout << "values: ";
  std::cout << h_values_out << std::endl;
}