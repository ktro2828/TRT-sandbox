#include <hipcub/hipcub.hpp>

#include <iostream>
#include <vector>

/**
 * hipcub::DeviceRadixSort::SortPairs(...)
 *
 * Sorts key-value pairs into ascensing order. (~2N auxiliary storage required)
 */

template <typename T>
std::ostream & operator<<(std::ostream & os, const std::vector<T> & v)
{
  os << "[";
  for (const auto & e : v) {
    os << e << ", ";
  }
  os << "]";
  return os;
}

int main()
{
  std::vector<float> h_keys{8.0f, 6.0f, 7.0f, 5.0f, 3.0f, 0.0f, 9.0f};
  std::vector<float> h_values{0.0f, 1.0f, 2.0f, 3.0f, 4.0f, 5.0, 6.0f};

  std::cout << "[Before]:\n";
  std::cout << "keys: ";
  std::cout << h_keys << std::endl;
  std::cout << "values: ";
  std::cout << h_values << std::endl;

  size_t num_items = h_keys.size();
  float *d_keys_in, *d_keys_out;
  float *d_values_in, *d_values_out;

  hipMalloc(&d_keys_in, sizeof(float) * num_items);
  hipMalloc(&d_keys_out, sizeof(float) * num_items);
  hipMalloc(&d_values_in, sizeof(float) * num_items);
  hipMalloc(&d_values_out, sizeof(float) * num_items);

  hipMemcpy(d_keys_in, h_keys.data(), sizeof(float) * num_items, hipMemcpyHostToDevice);
  hipMemcpy(d_values_in, h_values.data(), sizeof(float) * num_items, hipMemcpyHostToDevice);

  // Determinate temporary device storage requirements
  void * d_temp_storage{nullptr};
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(
    d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out,
    num_items);

  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(
    d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out,
    num_items);

  std::vector<float> h_keys_out(num_items);
  std::vector<float> h_values_out(num_items);
  hipMemcpy(h_keys_out.data(), d_keys_out, sizeof(float) * num_items, hipMemcpyDeviceToHost);
  hipMemcpy(h_values_out.data(), d_values_out, sizeof(float) * num_items, hipMemcpyDeviceToHost);

  // keys: [0, 3, 5, 6, 7, 8, 9]
  // values: [5, 4, 3, 1, 2, 0, 6]
  std::cout << "[After]:\n";
  std::cout << "keys: ";
  std::cout << h_keys_out << std::endl;
  std::cout << "values: ";
  std::cout << h_values_out << std::endl;
}