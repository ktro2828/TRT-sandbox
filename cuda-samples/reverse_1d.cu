
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void reverse(float * d_a, float * d_b, const size_t N)
{
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  d_b[N - idx - 1] = d_a[idx];
}

int main()
{
  constexpr size_t N = 256;
  float *h_a, *h_b;
  float *d_a, *d_b;

  h_a = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  h_b = reinterpret_cast<float *>(malloc(sizeof(float) * N));

  for (size_t i = 0; i < N; ++i) {
    h_a[i] = static_cast<float>(i);
  }

  hipMalloc(reinterpret_cast<void **>(&d_a), sizeof(float) * N);
  hipMalloc(reinterpret_cast<void **>(&d_b), sizeof(float) * N);

  hipMemcpy(d_a, h_a, sizeof(float) * N, hipMemcpyHostToDevice);
  reverse<<<1, N>>>(d_a, d_b, N);
  hipMemcpy(h_b, d_b, sizeof(float) * N, hipMemcpyDeviceToHost);

  std::cout << "(";
  for (size_t i = 0; i < N; ++i) {
    std::cout << h_b[i] << ", ";
  }
  std::cout << ")" << std::endl;

  hipFree(d_a);
  hipFree(d_b);
}