
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

// NOTE: Maximum N is 1024
constexpr int N = 1024;

__global__ void addVec(float * out, float * a, float * b)
{
  int i = threadIdx.x;
  out[i] = a[i] + b[i];
}

__host__ void printVec(float * v)
{
  std::cout << "(";
  for (int i = 0; i < N; ++i) {
    std::cout << v[i];
    if (i != N - 1) {
      std::cout << ", ";
    }
  }
  std::cout << ")";
}

int main()
{
  float *a, *b, *out;
  float *d_a, *d_b, *d_out;

  // Allocate memory
  a = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  b = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  out = reinterpret_cast<float *>(malloc(sizeof(float) * N));

  // Initialize array
  for (int i = 0; i < N; ++i) {
    a[i] = 1.0f;
    b[i] = 2.0f;
  }

  // Allocate device memory
  hipMalloc(reinterpret_cast<void **>(&d_a), sizeof(float) * N);
  hipMalloc(reinterpret_cast<void **>(&d_b), sizeof(float) * N);
  hipMalloc(reinterpret_cast<void **>(&d_out), sizeof(float) * N);

  // Transfer data from host to device memory
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

  // Executing kernel
  std::chrono::system_clock::time_point start, end;
  start = std::chrono::system_clock::now();
  dim3 threadsPerBlock(N);
  dim3 numBlocks(N / threadsPerBlock.x);
  addVec<<<numBlocks, threadsPerBlock>>>(d_out, d_a, d_b);
  end = std::chrono::system_clock::now();
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "[Elapsed]: " << elapsed << " [ns]" << std::endl;

  // Transfer data back to host memory
  hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

  // Display result
  printVec(out);

  // Deallocate device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);

  // Deallocate host memory
  free(a);
  free(b);
  free(out);
}